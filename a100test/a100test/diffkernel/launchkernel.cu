#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <pthread.h>
#include "hip/hip_runtime.h"

#define CHECK(func) \
{   \
    if (func != hipSuccess) { \
        printf("Function Call %s Failed!\n", #func);    \
        exit(1);    \
    }   \
}

#define THREAD_NUM  (1)

hipDevice_t device;
hipCtx_t context;
hipModule_t module;
hipFunction_t function;
pthread_mutex_t lock;
uint64_t count = 0;

void initCUDA(void)
{
    int devCount = 0;

    CHECK(hipInit(0));
    CHECK(hipGetDeviceCount(&devCount));

    if (devCount == 0) {
        printf("No Device!\n");
        exit(1);
    }

    CHECK(hipDeviceGet(&device, 0));
    CHECK(hipCtxCreate(&context, 0, device));

    CHECK(hipModuleLoad(&module, "kernel.ptx"));
    CHECK(hipModuleGetFunction(&function, module, "TestKernel"));
}

void __HostFunc(void *data)
{
    // just loop
    while (1) {
        sleep(1);
    }
}

uint32_t tid = 0;
void *ThreadFunc(void *data)
{
    hipStream_t stream;
    hipDeviceptr_t d_a, d_b;
    void *h_a;
    uint32_t ltid = __sync_fetch_and_add(&tid, 1);
    uint32_t lcount = 0;

    CHECK(hipCtxSetCurrent(context));
    for (;;) {
        CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        printf("stream number is %d\n", lcount++);
    }

    return NULL;
    CHECK(hipMalloc(&d_a, 1024));
    CHECK(hipMalloc(&d_b, 1024));
    CHECK(hipMemAllocHost((void **)&h_a, 1024));

//    CHECK(hipLaunchHostFunc(stream, __HostFunc, NULL));
    for (;;) {
        CHECK(hipModuleLaunchKernel(function,
            1, 1, 1,
            1, 1, 1,
            0, stream, NULL, NULL));

//        CHECK(hipLaunchHostFunc(stream, __HostFunc, NULL));
        CHECK(hipMemcpyHtoDAsync(d_a, h_a, 1024, stream));

        pthread_mutex_lock(&lock);
        printf("[%d] count = %ld, local count = %d\n",
                ltid, __sync_add_and_fetch(&count, 1),
                lcount++);
        pthread_mutex_unlock(&lock);
    }

    return NULL;
}

int main(int argc, char **argv)
{
    pthread_t _id[THREAD_NUM] = {0};

    printf("- Init...\n");
    initCUDA();

    pthread_mutex_init(&lock, 0);

    for (int i = 0; i < THREAD_NUM; i++) {
        CHECK(pthread_create(&_id[i], NULL, ThreadFunc, NULL));
    }

    for (int i = 0; i < THREAD_NUM; i++) {
        pthread_join(_id[i], NULL);
    }

//    CHECK(hipStreamSynchronize(0));

    return 0;
}
