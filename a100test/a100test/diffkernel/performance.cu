#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <pthread.h>
#include <sys/time.h>
#include "math.h"
#include "hip/hip_runtime.h"

#define CHECK(func) \
{   \
    if (func != hipSuccess) { \
        printf("Function Call %s Failed!\n", #func);    \
        exit(1);    \
    }   \
}

/********************************/
float Avg(float *Array, int Length)
{
    float sum = 0;
    for (int i = 0; i < Length; i++) {
        sum += Array[i];
    }

    return sum/Length;
}

float Std(float *Array, int Length)
{
    double var, avg;
    avg = Avg(Array, Length);
    for (int i = 0; i < Length; i++) {
        var += pow(Array[i] - avg, 2)/Length;
    }
    return pow(var, 0.5);
}

float Max(float *Array, int Length)
{
    float max = 0;
    for (int i = 0; i < Length; i++) {
        if (max < Array[i])
            max = Array[i];
    }
    return max;
}

float Min(float *Array, int Length)
{
    float min = 999999999999.0;
    for (int i = 0; i < Length; i++) {
        if (min > Array[i])
            min = Array[i];
    }
    return min;
}

/********************************/
hipDevice_t device;
hipCtx_t context;
hipModule_t module[1000];
hipFunction_t fast[1000], slow, calcu;
float *ResArray;

void initCUDA(void)
{
    int devCount = 0;

    CHECK(hipInit(0));
    CHECK(hipGetDeviceCount(&devCount));

    if (devCount == 0) {
        printf("No Device!\n");
        exit(1);
    }

    CHECK(hipDeviceGet(&device, 0));
    CHECK(hipCtxCreate(&context, 0, device));

    if (cuCtxResetPersistingL2Cache()) {
        printf("Use Cuda ToolKit Version 10.x\n");
    } else {
        printf("Use Cuda ToolKit Version 11.x\n");
    }

for (int i = 0; i < 1000; i++) {
    CHECK(hipModuleLoad(&module[i], "test.ptx"));
    CHECK(hipModuleGetFunction(&fast[i], module[i], "FastKernel"));
}
    CHECK(hipModuleGetFunction(&slow, module[0], "SlowKernel"));
    CHECK(hipModuleGetFunction(&calcu, module[0], "CalcuKernel"));
}

void exitCUDA(void)
{

for (int i = 0; i < 1000; i++) {
    CHECK(hipModuleUnload(module[i]));
}
    CHECK(hipCtxDestroy(context));
}

void PipeLineTime(int cycle_num)
{
    hipStream_t stream;
    hipDeviceptr_t d_a;
    struct timeval start, end;
    float sw_time = 0;
    void *params[] = {&d_a};

    CHECK(hipCtxSetCurrent(context));
    CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    CHECK(hipMalloc(&d_a, 1024));

    gettimeofday(&start, NULL);
    for (int i = 0; i < cycle_num; i++) {
            CHECK(hipModuleLaunchKernel(fast[0],
                1, 1, 1,
                1, 1, 1,
                0, stream, params, NULL));
    }

    CHECK(hipStreamSynchronize(stream));
    gettimeofday(&end, NULL);

    sw_time = (end.tv_sec * 1000000 + end.tv_usec) -
        (start.tv_sec * 1000000 + start.tv_usec);

    printf("total time = %f us, single time = %f us\n", sw_time, sw_time/cycle_num);

    gettimeofday(&start, NULL);
    for (int i = 0; i < cycle_num; i++) {
            CHECK(hipModuleLaunchKernel(fast[cycle_num % 1000],
                1, 1, 1,
                1, 1, 1,
                0, stream, params, NULL));
    }

    CHECK(hipStreamSynchronize(stream));
    gettimeofday(&end, NULL);

    sw_time = (end.tv_sec * 1000000 + end.tv_usec) -
        (start.tv_sec * 1000000 + start.tv_usec);

    printf("diff kernel total time = %f us, single time = %f us\n", sw_time, sw_time/cycle_num);
}

int main(int argc, char **argv)
{
    ResArray = (float *)calloc(8192, sizeof(float));
    if (!ResArray) {
        printf("Malloc Failed!\n");
        return 1;
    }

    printf("- Init...\n");
    initCUDA();

    PipeLineTime(8192);
    //LaunchDelay(8192);

    exitCUDA();
    return 0;
}
