#include <stdio.h>
#include <pthread.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define CHECK(func) \
{   \
    if (func != hipSuccess) { \
        printf("Function Call %s Failed!\n", #func);    \
        exit(1);    \
    }   \
}

#define SIZE		(20UL << 20)
hipDeviceptr_t data1, data2;
hipDevice_t device;
hipCtx_t context1, context2;
hipModule_t module1, module2;
hipFunction_t function1, function2;

volatile int reset_flag = 0;
volatile int done_flag = 0;
volatile int seq_flag = 0;

#define LAUNCH(func)	\
{	\
	gettimeofday(&start, NULL);	\
        CHECK(hipModuleLaunchKernel(func,	\
            1, 1, 1,	\
            1, 1, 1,	\
            0, stream, params, NULL));	\
   	CHECK(hipStreamSynchronize(stream));	\
	gettimeofday(&end, NULL);	\
	\
	sw_time = (end.tv_sec * 1000000 + end.tv_usec) -	\
		(start.tv_sec * 1000000 + start.tv_usec);	\
	printf("%d l2 cache sw_time = %f\n", tid, sw_time);	\
}

void initCUDA(void)
{
    int devCount = 0;

    CHECK(hipInit(0));
    CHECK(hipGetDeviceCount(&devCount));

    if (devCount == 0) {
        printf("No Device!\n");
        exit(1);
    }

    CHECK(hipDeviceGet(&device, 0));
    CHECK(hipCtxCreate(&context1, 0, device));
    CHECK(hipModuleLoad(&module1, "kernel.ptx"));
    CHECK(hipModuleGetFunction(&function1, module1, "TestKernel"));

    CHECK(hipCtxCreate(&context2, 0, device));
    CHECK(hipModuleLoad(&module2, "kernel.ptx"));
    CHECK(hipModuleGetFunction(&function2, module2, "TestKernel"));
}

void *ThreadFunc1(void *data)
{
	hipStream_t stream;
	struct timeval start, end;
	float sw_time = 0;
	int size = SIZE;
	void *params[] = {&size, &data1};
	int tid = 1;

    	CHECK(hipCtxSetCurrent(context1));

    	CHECK(hipMalloc(&data1, SIZE));
        CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	/* set L2 persisting cache */
	hipLaunchAttributeValue attr;
	attr.accessPolicyWindow.base_ptr = (void *)data1;
	attr.accessPolicyWindow.num_bytes = SIZE;
	attr.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
	attr.accessPolicyWindow.hitRatio = 1;
	attr.accessPolicyWindow.missProp = hipAccessPropertyStreaming;

//	CHECK(hipStreamSetAttribute(stream,
//		hipLaunchAttributeAccessPolicyWindow, &attr));

        CHECK(hipModuleLaunchKernel(function1,
            1, 1, 1,
            1, 1, 1,
            0, stream, params, NULL));
   	CHECK(hipStreamSynchronize(stream));

	LAUNCH(function1);
	LAUNCH(function1);
	__sync_fetch_and_add(&seq_flag, 1);

	while (!__sync_fetch_and_add(&done_flag, 0))
		sleep(1);

	//cuCtxResetPersistingL2Cache();
   	CHECK(hipCtxSynchronize());
	printf("after reset\n");
	__sync_fetch_and_add(&reset_flag, 1);

	return NULL;
}

void *ThreadFunc2(void *data)
{
	hipStream_t stream;
	struct timeval start, end;
	float sw_time = 0;
	int size = SIZE;
	void *params[] = {&size, &data2};
	int tid = 2;

    	CHECK(hipCtxSetCurrent(context2));

    	CHECK(hipMalloc(&data2, SIZE));
        CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	/* set L2 persisting cache */
	hipLaunchAttributeValue attr;
	attr.accessPolicyWindow.base_ptr = (void *)data2;
	attr.accessPolicyWindow.num_bytes = SIZE;
	attr.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
	attr.accessPolicyWindow.hitRatio = 1;
	attr.accessPolicyWindow.missProp = hipAccessPropertyStreaming;

	CHECK(hipStreamSetAttribute(stream,
		hipLaunchAttributeAccessPolicyWindow, &attr));

	while (!__sync_fetch_and_add(&seq_flag, 0))
		sleep(1);

        CHECK(hipModuleLaunchKernel(function2,
            1, 1, 1,
            1, 1, 1,
            0, stream, params, NULL));
   	CHECK(hipStreamSynchronize(stream));

	LAUNCH(function2);
	LAUNCH(function2);

	__sync_fetch_and_add(&done_flag, 1);

	while (!__sync_fetch_and_add(&reset_flag, 0))
		sleep(1);

	/* after reset test */
	LAUNCH(function2);

	return NULL;
}

int main(int argc, char **argv)
{
	pthread_t id1, id2;

	printf("- Init...\n");
	initCUDA();

	CHECK(pthread_create(&id2, NULL, ThreadFunc2, NULL));
	CHECK(pthread_create(&id1, NULL, ThreadFunc1, NULL));

	CHECK(pthread_join(id1, NULL));
	CHECK(pthread_join(id2, NULL));

    return 0;
}
