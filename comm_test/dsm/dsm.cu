#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#define CUDA_CHECK(func) \
	do { \
		hipError_t ret = (func); \
		if (ret) { \
			printf("%s@%d call %s return %d\n", __func__, __LINE__, #func, ret); \
			exit(-1); \
		} \
	} while (0)

// Distributed Shared memory histogram kernel
__global__ void clusterHist_kernel(int *bins, const int nbins, const int bins_per_block, const int *__restrict__ input,
                                   size_t array_size)
{
  extern __shared__ int smem[];
  namespace cg = cooperative_groups;
  int tid = cg::this_grid().thread_rank();

  // Cluster initialization, size and calculating local bin offsets.
  cg::cluster_group cluster = cg::this_cluster();
  unsigned int clusterBlockRank = cluster.block_rank();
  int cluster_size = cluster.dim_blocks().x;

  for (int i = threadIdx.x; i < bins_per_block; i += blockDim.x)
  {
    smem[i] = 0; //Initialize shared memory histogram to zeros
  }

  // cluster synchronization ensures that shared memory is initialized to zero in
  // all thread blocks in the cluster. It also ensures that all thread blocks
  // have started executing and they exist concurrently.
  cluster.sync();

  for (int i = tid; i < array_size; i += blockDim.x * gridDim.x)
  {
    int ldata = input[i];

    //Find the right histogram bin.
    int binid = ldata;
    if (ldata < 0)
      binid = 0;
    else if (ldata >= nbins)
      binid = nbins - 1;

    //Find destination block rank and offset for computing
    //distributed shared memory histogram
    int dst_block_rank = (int)(binid / bins_per_block);
    int dst_offset = binid % bins_per_block;

    //Pointer to target block shared memory
    int *dst_smem = cluster.map_shared_rank(smem, dst_block_rank);

    //Perform atomic update of the histogram bin
    atomicAdd(dst_smem + dst_offset, 1);
  }

  // cluster synchronization is required to ensure all distributed shared
  // memory operations are completed and no thread block exits while
  // other thread blocks are still accessing distributed shared memory
  cluster.sync();

  // Perform global memory histogram, using the local distributed memory histogram
  int *lbins = bins + cluster.block_rank() * bins_per_block;
  for (int i = threadIdx.x; i < bins_per_block; i += blockDim.x)
  {
    atomicAdd(&lbins[i], smem[i]);
  }
}

int main()
{
  int *input;
  int array_size = 1024;
  int *bins;
  int nbins = 1024;
  hipMalloc(&input, array_size * sizeof(*input));
  hipMalloc(&bins, nbins * sizeof(*bins));
  hipLaunchConfig_t config = {0};
  int threads_per_block = 32;
  config.gridDim = array_size / threads_per_block;
  config.blockDim = threads_per_block;

  // cluster_size depends on the histogram size.
  // ( cluster_size == 1 ) implies no distributed shared memory, just thread block local shared memory
  int cluster_size = 16; // size 2 is an example here
  int nbins_per_block = nbins / cluster_size;

  //dynamic shared memory size is per block.
  //Distributed shared memory size =  cluster_size * nbins_per_block * sizeof(int)
  config.dynamicSmemBytes = nbins_per_block * sizeof(int);

  CUDA_CHECK(::hipFuncSetAttribute((void *)clusterHist_kernel, hipFuncAttributeMaxDynamicSharedMemorySize, config.dynamicSmemBytes));

  hipLaunchAttribute attribute[1];
  attribute[0].id = cudaLaunchAttributeClusterDimension;
  attribute[0].val.clusterDim.x = cluster_size;
  attribute[0].val.clusterDim.y = 1;
  attribute[0].val.clusterDim.z = 1;

  config.numAttrs = 1;
  config.attrs = attribute;

  cudaLaunchKernelEx(&config, clusterHist_kernel, bins, nbins, nbins_per_block, input, array_size);
  CUDA_CHECK(hipDeviceSynchronize());
}
